#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

void read_header(FILE* fp,int *numVars,int *numRest);
void read_data(FILE* fp,double *COEF,double *P,double *Z);
void print_matrix(double *data,int mr, int mc);
void print_array(double *data,int len);

__global__ void montecarlo(double *COEF,double *P,double *Z,double *C,double *CANS,int numRest, int numVars)
{ 
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    double z = 0,zans = 0;
    int i = 0,r = 0,j = 0;
    int max=1024;
    // x,y = x * numCol + y
    for(r=0;r<max;r++){
        while(i < numRest && r<max)
        {
            tmp = 0;
            for (j = 0; j < numVars; ++j)
            {   
                if (i == 0){
                    //si esta en la primera vuelta genera nuevos numeros
                    top =  P[i];
                    if (COEF[i*numVars+j]>0)
                    {
                        top = P[i]/COEF[i*numVars+j];
                    }
                    x = rand_in_range(0,top);
                    C[j] = x; 
                }
                tmp += C[j]*COEF[i*numVars+j];
            }
            if(tmp <= P[i]){
                //se cumple la condicion
                i++;
            }else{
                //si no se cumple alguna condicion
                //vuelve a empezar
                i = 0;
                r++;
            }
        }
        //si finaliza es porque se cumplieron todas las condiciones
        z=0;
        for (k = 0; k < numVars; ++k)
        {
            z+= C[k]*Z[k];
        }
        if (z>zans){
            for (k = 0; k < numVars; ++k)
            {
                CANS­[pos*(numVars+1)+k]=C[k];
            }
            CANS[pos*(numVars+1)+numVars] = z;
            zans = z;
        }
    }
}

int main(int argc, char **argv)
{
    if( 2 != argc ){
        printf("Incorrect number of params: %d\n",argc - 1);
        return 1;
    }
    //Vars Declaration
    int tile_dim = 1024,index = 0;
    int numRest,numVars,i = 0;
    double zans = 0,z = 0;
    double *gpu_COEF, *cpu_COEF;
    double *gpu_P, *cpu_P;
    double *gpu_Z, *cpu_Z;
    double *gpu_C, *cpu_C;
    double *gpu_CANS, *cpu_CANS;
    /*
    hipError_t err = hipSuccess;
    */
    // Open the files
    FILE* fp = fopen(argv[1], "r");
    // Check if files exists
    if (fp == NULL)
    {
        printf("Could not open file %s",argv[1] );
        return 1;
    }
    //Read the number of rows and cols of each matrix
    read_header(fp,&numVars,&numRest);
    printf("%i,%i\n", numVars,numRest);

    //reserve memory to each matrix
    int drv = numVars*numRest*sizeof(double);
    int dv = numVars*sizeof(double);
    int dr = numRest*sizeof(double);
    int cs = pow(tile_dim,2)*(numVars + 1)*sizeof(double);

    cpu_COEF = (double*)malloc(drv);
    cpu_P = (double*)malloc(dr);
    cpu_Z = (double*)malloc(dv);
    cpu_C = (double*)calloc(numVars+1,sizeof(double));
    cpu_CANS = (double*)calloc(pow(tile_dim,2)*(numVars + 1),sizeof(double));

    err = hipMalloc((void**)&gpu_COEF,drv);
    if(err != hipSuccess){printf("Error with COEF\n");exit(1);}
    err = hipMalloc((void**)&gpu_P,dr);
    if(err != hipSuccess){printf("Error with P\n");exit(1);}
    err = hipMalloc((void**)&gpu_Z,dv);
    if(err != hipSuccess){printf("Error with Z\n");exit(1);}
    err = hipMalloc((void**)&gpu_C,dv);
    if(err != hipSuccess){printf("Error with C\n");exit(1);}
    err = hipMalloc((void**)&gpu_CANS,cs);
    if(err != hipSuccess){printf("Error with CANS\n");exit(1);}

    //Read the Files to the CPU memory
    read_data(fp,cpu_COEF,cpu_P,cpu_Z);
    //Now the files can be closed
    fclose(fp);
    //Copy each matrix to the device
    err = hipMemcpy(gpu_COEF, cpu_COEF, drv, hipMemcpyHostToDevice);
    if(err != hipSuccess){printf("Error Coping COEF\n");exit(1);}
    err = hipMemcpy(gpu_P, cpu_P, dr, hipMemcpyHostToDevice);
    if(err != hipSuccess){printf("Error Coping P\n");exit(1);}
    err = hipMemcpy(gpu_Z, cpu_Z, dv, hipMemcpyHostToDevice);
    if(err != hipSuccess){printf("Error Coping Z\n");exit(1);}

    //Execute the Kernel in the Device
    dim3 block_dim(tile_dim,1,1);
    dim3 grid_dim(tile_dim,1,1);

    montecarlo<<<grid_dim,block_dim>>>(gpu_COEF,gpu_P,gpu_Z,gpu_C,gpu_CANS,numRest, numVars);

    //copy the result to Host mem
    err = hipMemcpy(cpu_CANS,gpu_CANS,cs, hipMemcpyDeviceToHost);
    if(err != hipSuccess){printf("Error Coping Matrix R\n");exit(1);}

    for (i = 0; i < tile_dim*tile_dim; ++i)
    {
        z = cpu_CANS[i*(numVars+1)+numVars];
        if(z < zans)­{
            zans = z;
            index = i;
        }
    }
    for (int i = 0; i < numVars; ++i)
    {
        C[i] = cpu_CANS[index*(numVars+1)+i];
    }  
    //to this point
    //zans = contains the greather Z
    //C contains the value of each var 

    free(cpu_COEF);
    free(cpu_P);
    free(cpu_Z);
    free(cpu_C);
    free(cpu_CANS);
}

void read_header(FILE* fp,int *numVars,int *numRest){
    char c;
    int nc = 0;
    int temp = 0;
    int pot = 0;
	for (c = getc(fp); c != '\n'; c = getc(fp)){
        nc = (int)c;
        if(nc >= 48 && nc <= 57){
            //add other decimal
            temp = (temp * (int)pow(10,pot)) + (nc - 48);
            if(pot == 0){
                pot = 1;
            }
        }
        if (c == ','){
            (*numVars) = temp;
            temp = 0;
            pot = 0;
        }
    } 
    (*numRest) = temp;
    rewind(fp);
}
void read_data(FILE* fp,double *COEF,double *P,double *Z){
    char c;
    int nc,numVars,numRest;
    int temp = 0;
    int pot = 0;
    int index = 0;
    //read vars & rest
    for (c = getc(fp); c != '\n'; c = getc(fp)){
        nc = (int)c;
        if(nc >= 48 && nc <= 57){
            //add other decimal
            temp = (temp * (int)pow(10,pot)) + (nc - 48);
            if(pot == 0){
                pot = 1;
            }
        }
        if (c == ','){
            numVars = temp;
            temp = 0;
            pot = 0;
        }
    }
    numRest = temp;
    temp = 0;
    pot = 0;
    //read the COEF
    for (c = getc(fp); index < numRest*numVars; c = getc(fp)){
        nc = (int)c;
        if (c == ',' || c == '\n') {
            //save & restart
            COEF[index] = (double)temp;
            temp = 0;
            pot = 0;
            index++;
        }else if(nc >= 48 && nc <= 57){
            //add other decimal
            temp = (temp * (int)pow(10,pot)) + (nc - 48);
            if(pot == 0){
                pot = 1;
            }
        }
    }
    //read P
    nc = (int)c;
    index = 0;
    temp = nc - 48;
    pot = 1;
    //read the COEF
    for (c = getc(fp); index < numRest; c = getc(fp)){
        nc = (int)c;
        if (c == ',' || c == '\n') {
            //save & restart
            P[index] = (double)temp;
            temp = 0;
            pot = 0;
            index++;
        }else if(nc >= 48 && nc <= 57){
            //add other decimal
            temp = (temp * (int)pow(10,pot)) + (nc - 48);
            if(pot == 0){
                pot = 1;
            }
        }
    }
    //read Z
    nc = (int)c;
    index = 0;
    temp = nc - 48;
    pot = 1;
    //read the COEF
    for (c = getc(fp); c!=EOF && index<numVars; c = getc(fp)){
        nc = (int)c;
        if (c == ',' || c == '\n') {
            //save & restart
            Z[index] = (double)temp;
            temp = 0;
            pot = 0;
            index++;
        }else if(nc >= 48 && nc <= 57){
            //add other decimal
            temp = (temp * (int)pow(10,pot)) + (nc - 48);
            if(pot == 0){
                pot = 1;
            }
        }
    }

    rewind(fp);
    return;
}
void print_matrix(double *data,int mRr, int mRc){
    int row, col;
    for (row=0; row<mRr; row++)
    {
        for(col=0; col<mRc; col++)
            {
             printf("%g", data[row * mRc + col]);
	     if(col != mRc - 1){
		printf(",");
	     }
            }
        if(row != mRr - 1){
            //if isn't the last line print return
            printf("\n");
        }
    }
    printf("\n");
    return;
}
void print_array(double *data,int len){
    int i;
    for (i=0; i<len; i++)
    {
        printf("%g", data[i]);
        if(i != len - 1){
            printf(",");
        }
    }
    printf("\n");
    return;
}
