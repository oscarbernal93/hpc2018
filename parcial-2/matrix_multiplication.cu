
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

int row_counter(FILE* fp);
int col_counter(FILE* fp);
void read_matrix(FILE* fp,int *data);
void print_matrix(int *data,int mRr, int mRc);

__global__ void matrix_multiplication(int *m1,int *m2, int *mR, int m1r, int m1c, int m2c)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    int i = 0;
    if( col < m2c && row < m1r) 
    {
        for(i = 0; i < m1c; i++) 
        {
            sum += m1[row * m1c + i] * m2[i * m2c + col];
        }
        mR[row * m2c + col] = sum;
    }
} 

int main(int argc, char **argv)
{
    if( 3 != argc ){
        printf("Incorrect number of params: %d\n",argc - 1);
        return 1;
    }
    //Vars Declaration
    int m1r,m1c,m2r,m2c,mRr,mRc; //rows & cols
    int m1s,m2s,mRs;  //size
    int *gpu_m1, *cpu_m1;
    int *gpu_m2, *cpu_m2;
    int *gpu_mR, *cpu_mR;
    hipError_t err = hipSuccess;
	// Open the files
    FILE* fp1 = fopen(argv[1], "r");
    FILE* fp2 = fopen(argv[2], "r");
    // Check if files exists
    if (fp1 == NULL)
    {
        printf("Could not open file %s",argv[1] );
        return 1;
    }
    if (fp2 == NULL)
    {
        printf("Could not open file %s",argv[2] );
        return 1;
    }
    //Read the number of rows and cols of each matrix
    m1r = row_counter(fp1);
    m1c = col_counter(fp1);
    m2r = row_counter(fp2);
    m2c = col_counter(fp2);
    mRr = m1r;
    mRc = m2c;
    //printf("Matrix 1: %dx%d\n", m1r,m1c);
    //printf("Matrix 2: %dx%d\n", m2r,m2c);
	
    //condition: the number of columns in A must equal the number of rows in B
    if(m1c != m2r){
        printf("Incorrect Matrix Size, cols of Matrix 1 (%d) are different of the rows of Matrix 2 (%d)\n",m1c,m2r );
        return 1;
    }

    //reserve memory to each matrix
    m1s = m1r*m1c*sizeof(int);
    m2s = m2r*m2c*sizeof(int);
    mRs = mRr*mRc*sizeof(int);
    cpu_m1 = (int*)malloc(m1s);
    cpu_m2 = (int*)malloc(m2s);
    cpu_mR = (int*)malloc(mRs);
    err = hipMalloc((void**)&gpu_m1,m1s);
    if(err != hipSuccess){printf("Error with Matrix 1\n");exit(1);}
    err = hipMalloc((void**)&gpu_m2,m2s);
    if(err != hipSuccess){printf("Error with Matrix 2\n");exit(1);}
    err = hipMalloc((void**)&gpu_mR,mRs);
    if(err != hipSuccess){printf("Error with Matrix R\n");exit(1);}

    //Read the Files to the CPU memory
    read_matrix(fp1,cpu_m1);
    read_matrix(fp2,cpu_m2);
    //Now the files can be closed
    fclose(fp1);
	fclose(fp2);

    //Copy each matrix to the device
    err = hipMemcpy(gpu_m1, cpu_m1, m1s, hipMemcpyHostToDevice);
    if(err != hipSuccess){printf("Error Coping Matrix 1\n");exit(1);}
    err = hipMemcpy(gpu_m2, cpu_m2, m2s, hipMemcpyHostToDevice);
    if(err != hipSuccess){printf("Error Coping Matrix 2\n");exit(1);}

    //Execute the Kernel in the Device
    int tile_dim = 32;
    dim3 block_dim(tile_dim,tile_dim,1);
    dim3 grid_dim(ceil(mRc/float(tile_dim)),ceil(mRr/float(tile_dim)),1);

    matrix_multiplication<<<grid_dim,block_dim>>>(gpu_m1, gpu_m2, gpu_mR, m1r, m1c, m2c);

    //copy the result to Host mem
    err = hipMemcpy(cpu_mR,gpu_mR,mRs, hipMemcpyDeviceToHost);
    if(err != hipSuccess){printf("Error Coping Matrix R\n");exit(1);}

    print_matrix(cpu_mR, mRr, mRc);

}

int row_counter(FILE* fp){
    char c;
	int count = 1;
	for (c = getc(fp); c != EOF; c = getc(fp))
        if (c == '\n') 
            count = count + 1;
	rewind(fp);
	return count;
}
int col_counter(FILE* fp){
    char c;
    int count = 1;
    for (c = getc(fp); c != '\n'; c = getc(fp))
        if (c == ',') 
            count = count + 1;
    rewind(fp);
    return count;
}
void read_matrix(FILE* fp,int *data){
    char c;
    int nc;
    int temp = 0;
    int pot = 0;
    int index = 0;
    for (c = getc(fp); c != EOF; c = getc(fp)){
        nc = (int)c;
        if (c == ',' || c == '\n') {
            //save & restart
            data[index] = (int)temp;
            temp = 0;
            pot = 0;
            index++;
        }else if(nc >= 48 && nc <= 57){
            //add other decimal
            temp = (temp * (int)pow(10,pot)) + (nc - 48);
            if(pot == 0){
                pot = 1;
            }
        }
    }
    rewind(fp);
    return;
}
void print_matrix(int *data,int mRr, int mRc){
    int row, col;
    for (row=0; row<mRr; row++)
    {
        for(col=0; col<mRc; col++)
            {
             printf("%d", data[row * mRc + col]);
	     if(col != mRc - 1){
		printf(",");
	     }
            }
        if(row != mRr - 1){
            //if isn't the last line print return
            printf("\n");
        }
    }
    return;
}
