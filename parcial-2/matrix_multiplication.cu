
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

int row_counter(FILE* fp);
int col_counter(FILE* fp);
void read_matrix(FILE* fp,unsigned int *data);

int main(int argc, char **argv)
{
    if( 3 != argc ){
        printf("Incorrect number of params: %d\n",argc - 1);
        return 1;
    }
    //Vars Declaration
    int m1r,m1c,m2r,m2c,mRr,mRc;
    unsigned int *gpu_m1, *cpu_m1;
    unsigned int *gpu_m2, *cpu_m2;
    unsigned int *gpu_mR, *cpu_mR;
    /// cudaError_t err = cudaSuccess;
	// Open the files
    FILE* fp1 = fopen(argv[1], "r");
    FILE* fp2 = fopen(argv[2], "r");
    // Check if files exists
    if (fp1 == NULL)
    {
        printf("Could not open file %s",argv[1] );
        return 1;
    }
    if (fp2 == NULL)
    {
        printf("Could not open file %s",argv[2] );
        return 1;
    }
    //Read the number of rows and cols of each matrix
    m1r = row_counter(fp1);
    m1c = col_counter(fp1);
    m2r = row_counter(fp2);
    m2c = col_counter(fp2);
    mRr = m1r;
    mRc = m2c;
    printf("Matrix 1: %dx%d\n", m1r,m1c);
    printf("Matrix 2: %dx%d\n", m2r,m2c);
 	
    //condition: the number of columns in A must equal the number of rows in B
    if(m1c != m2r){
        printf("Incorrect Matrix Size, cols of Matrix 1 (%d) are different of the rows of Matrix 2 (%d)\n",m1c,m2r );
        return 1;
    }

    //reserve memory to each matrix
    cpu_m1 = (unsigned int*)malloc(m1r*m1c*sizeof(unsigned int));
    cpu_m2 = (unsigned int*)malloc(m2r*m2c*sizeof(unsigned int));
    cpu_mR = (unsigned int*)malloc(mRr*mRc*sizeof(unsigned int));
    /// err = cudaMalloc((void**)&gpu_m1,m1r*m1c*sizeof(unsigned int));
    /// if(err != cudaSuccess){printf("Error with Matrix 1\n");exit(1);}
    /// err = cudaMalloc((void**)&gpu_m2,m2r*m2c*sizeof(unsigned int));
    /// if(err != cudaSuccess){printf("Error with Matrix 2\n");exit(1);}
    /// err = cudaMalloc((void**)&gpu_mR,mRr*mRc*sizeof(unsigned int));
    /// if(err != cudaSuccess){printf("Error with Matrix R\n");exit(1);}

    //Read the Files to the CPU memory
    read_matrix(fp1,cpu_m1);
    read_matrix(fp2,cpu_m2);
    //Now the files can be closed
    fclose(fp1);
 	fclose(fp2);

}

int row_counter(FILE* fp){
    char c;
	int count = 1;
	for (c = getc(fp); c != EOF; c = getc(fp))
        if (c == '\n') 
            count = count + 1;
	rewind(fp);
	return count;
}
int col_counter(FILE* fp){
    char c;
    int count = 1;
    for (c = getc(fp); c != '\n'; c = getc(fp))
        if (c == ',') 
            count = count + 1;
    rewind(fp);
    return count;
}
void read_matrix(FILE* fp,unsigned int *data){
    char c;
    int nc;
    unsigned int temp = 0;
    int pot = 0;
    int index = 0;
    for (c = getc(fp); c != EOF; c = getc(fp)){
        nc = (int)c;
        if (c == ',' || c == '\n') {
            //save & restart
            data[index] = (unsigned int)temp;
            temp = 0;
            pot = 0;
            index++;
        }else if(nc >= 48 && nc <= 57){
            //add other decimal
            temp = (temp * (int)pow(10,pot)) + (nc - 48);
            if(pot == 0){
                pot = 1;
            }
            printf("=%d\n",temp );
        }
    }
    rewind(fp);
    return;
}